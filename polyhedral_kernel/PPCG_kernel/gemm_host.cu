#include <assert.h>
#include <stdio.h>
#include "gemm_kernel.hu"
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <sys/time.h>
#include <omp.h>

#include <hip/hip_runtime.h>
#include <iostream>

#define Smax 1
#define Smin 0
#define alpha 0.6

void Init_Matrix_rand_PSxM(float matrix[1024][1024], int PS1, int PS2) {
    for (int i = 0; i < PS1; i++) {
        for (int j = 0; j < PS2; j++) {
            matrix[i][j] = (float)rand() / 32767 * (Smax - Smin) + Smin;
        }
    }
}
void Init_array_rand_PSxM(float matrix[1024], int PS1) {
    for (int i = 0; i < PS1; i++) {
        matrix[i] = (float)rand() / 32767 * (Smax - Smin) + Smin;
    }
}

void Init_Matrix_zero_PSxM(float matrix[1024][1024], int PS1, int PS2) {
    for (int i = 0; i < PS1; i++) {
        for (int j = 0; j < PS2; j++) {
            matrix[i][j] = 0;
        }
    }
}

void Init_array_zero_PSxM(float matrix[1024], int PS1) {
    for (int i = 0; i < PS1; i++) {
        matrix[i] = 0;
    }
}



int main(){
    float A[1024][1024], B[1024][1024],C[1024][1024];
    double Total_time = 0.0;
    srand((unsigned)time(NULL));
    
    Init_Matrix_rand_PSxM(A,1024,1024);
    Init_Matrix_rand_PSxM(B,1024,1024);
    Init_Matrix_rand_PSxM(C,1024,1024);
    
    
    struct timespec start, finish;
    double elapsed;
    
    float milliseconds = 0;
    clock_gettime(CLOCK_MONOTONIC, &start);



  {
#define cudaCheckReturn(ret) \
  do { \
    hipError_t cudaCheckReturn_e = (ret); \
    if (cudaCheckReturn_e != hipSuccess) { \
      fprintf(stderr, "CUDA error: %s\n", hipGetErrorString(cudaCheckReturn_e)); \
      fflush(stderr); \
    } \
    assert(cudaCheckReturn_e == hipSuccess); \
  } while(0)
#define cudaCheckKernel() \
  do { \
    cudaCheckReturn(hipGetLastError()); \
  } while(0)

    float *dev_A;
    float *dev_B;
    float *dev_C;
    
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    
    cudaCheckReturn(hipMalloc((void **) &dev_A, (1024) * (1024) * sizeof(float)));
    cudaCheckReturn(hipMalloc((void **) &dev_B, (1024) * (1024) * sizeof(float)));
    cudaCheckReturn(hipMalloc((void **) &dev_C, (1024) * (1024) * sizeof(float)));
    
    cudaCheckReturn(hipMemcpy(dev_A, A, (1024) * (1024) * sizeof(float), hipMemcpyHostToDevice));
    cudaCheckReturn(hipMemcpy(dev_B, B, (1024) * (1024) * sizeof(float), hipMemcpyHostToDevice));
    cudaCheckReturn(hipMemcpy(dev_C, C, (1024) * (1024) * sizeof(float), hipMemcpyHostToDevice));
    {
      dim3 k0_dimBlock(16, 32);
      dim3 k0_dimGrid(32, 32);
      hipEventRecord(start,0);
      kernel0 <<<k0_dimGrid, k0_dimBlock>>> (dev_A, dev_B, dev_C);
      hipEventRecord(stop,0);
      hipEventSynchronize(stop);
      cudaCheckKernel();
      hipEventElapsedTime(&milliseconds, start, stop);
    }
    
    cudaCheckReturn(hipMemcpy(C, dev_C, (1024) * (1024) * sizeof(float), hipMemcpyDeviceToHost));
    cudaCheckReturn(hipFree(dev_A));
    cudaCheckReturn(hipFree(dev_B));
    cudaCheckReturn(hipFree(dev_C));
  }


    clock_gettime(CLOCK_MONOTONIC, &finish);
    elapsed = (finish.tv_sec - start.tv_sec);
    elapsed += (finish.tv_nsec - start.tv_nsec) / 1000000000.0;
    printf("%.10f\n",elapsed);
    printf("%.10f\n", milliseconds/1000);
    return 0;
}




