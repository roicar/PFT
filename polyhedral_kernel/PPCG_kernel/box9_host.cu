#include <assert.h>
#include <stdio.h>
#include "seidel_kernel.hu"
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <sys/time.h>
#include <omp.h>

#include <hip/hip_runtime.h>
#include <iostream>
#define Smax 1
#define Smin 0
#define alpha 0.6

void Init_Matrix_rand_PSxM(float matrix[1024][1024], int PS1, int PS2) {
    for (int i = 0; i < PS1; i++) {
        for (int j = 0; j < PS2; j++) {
            matrix[i][j] = (float)rand() / 32767 * (Smax - Smin) + Smin;
        }
    }
}

void Init_Matrix_zero_PSxM(float matrix[1024][1024], int PS1, int PS2) {
    for (int i = 0; i < PS1; i++) {
        for (int j = 0; j < PS2; j++) {
            matrix[i][j] = 0;
        }
    }
}

int main() {
    int i, j;
    float a[1024][1024],b[1024][1024];
    srand((unsigned)time(NULL));

    Init_Matrix_rand_PSxM(a, 1024, 1024);
    Init_Matrix_zero_PSxM(b,1024,1024);
    struct timespec start, finish;
    double elapsed;
    float milliseconds = 0;
    clock_gettime(CLOCK_MONOTONIC, &start);

    {
#define cudaCheckReturn(ret) \
  do { \
    hipError_t cudaCheckReturn_e = (ret); \
    if (cudaCheckReturn_e != hipSuccess) { \
      fprintf(stderr, "CUDA error: %s\n", hipGetErrorString(cudaCheckReturn_e)); \
      fflush(stderr); \
    } \
    assert(cudaCheckReturn_e == hipSuccess); \
  } while(0)
#define cudaCheckKernel() \
  do { \
    cudaCheckReturn(hipGetLastError()); \
  } while(0)

      float *dev_a;
      float *dev_b;
      hipEvent_t start, stop;
      hipEventCreate(&start);
      hipEventCreate(&stop);
      
      cudaCheckReturn(hipMalloc((void **) &dev_a, (1024) * (1024) * sizeof(float)));
      cudaCheckReturn(hipMalloc((void **) &dev_b, (1023) * (1024) * sizeof(float)));
      
      cudaCheckReturn(hipMemcpy(dev_a, a, (1024) * (1024) * sizeof(float), hipMemcpyHostToDevice));
      cudaCheckReturn(hipMemcpy(dev_b, b, (1023) * (1024) * sizeof(float), hipMemcpyHostToDevice));
      {
        dim3 k0_dimBlock(16, 32);
        dim3 k0_dimGrid(32, 32);
        hipEventRecord(start,0);
        kernel0 <<<k0_dimGrid, k0_dimBlock>>> (dev_a, dev_b);
        hipEventRecord(stop,0);
        hipEventSynchronize(stop);
        cudaCheckKernel();
        hipEventElapsedTime(&milliseconds, start, stop);
      }
      
      cudaCheckReturn(hipMemcpy(b, dev_b, (1023) * (1024) * sizeof(float), hipMemcpyDeviceToHost));
      cudaCheckReturn(hipFree(dev_a));
      cudaCheckReturn(hipFree(dev_b));
    }

    clock_gettime(CLOCK_MONOTONIC, &finish);
    elapsed = (finish.tv_sec - start.tv_sec);
    elapsed += (finish.tv_nsec - start.tv_nsec) / 1000000000.0;
    printf("%.10f\n", elapsed);
    printf("%.10f\n", milliseconds/1000);
    return 0;
}

