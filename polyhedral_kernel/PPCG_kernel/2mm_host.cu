#include <assert.h>
#include <stdio.h>
#include "2mm_kernel.hu"
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <sys/time.h>
#include <omp.h>
#include <hip/hip_runtime.h>
#include <iostream>

#define Smax 1
#define Smin 0
#define alpha 0.6


void Init_Matrix_rand_PSxM(float matrix[1024][1024], int PS1, int PS2) {
    for (int i = 0; i < PS1; i++) {
        for (int j = 0; j < PS2; j++) {
            matrix[i][j] = (float)rand() / 32767 * (Smax - Smin) + Smin;
        }
    }
}

void Init_Matrix_zero_PSxM(float matrix[1024][1024], int PS1, int PS2) {
    for (int i = 0; i < PS1; i++) {
        for (int j = 0; j < PS2; j++) {
            matrix[i][j] = 0;
        }
    }
}

int main() {
    float A[1024][1024], B[1024][1024], C[1024][1024], D[1024][1024], E[1024][1024];
    double Total_time = 0.0;
    srand((unsigned)time(NULL));
    Init_Matrix_zero_PSxM(C, 1024,1024);
    Init_Matrix_rand_PSxM(A, 1024,1024);
    Init_Matrix_rand_PSxM(B, 1024,1024);
    Init_Matrix_rand_PSxM(D, 1024,1024);
    Init_Matrix_rand_PSxM(E, 1024,1024);
    struct timespec start, finish;
    double elapsed;
    float milliseconds = 0;
    clock_gettime(CLOCK_MONOTONIC, &start);
    {
#define cudaCheckReturn(ret) \
  do { \
    hipError_t cudaCheckReturn_e = (ret); \
    if (cudaCheckReturn_e != hipSuccess) { \
      fprintf(stderr, "CUDA error: %s\n", hipGetErrorString(cudaCheckReturn_e)); \
      fflush(stderr); \
    } \
    assert(cudaCheckReturn_e == hipSuccess); \
  } while(0)
#define cudaCheckKernel() \
  do { \
    cudaCheckReturn(hipGetLastError()); \
  } while(0)

      float *dev_A;
      float *dev_B;
      float *dev_C;
      float *dev_D;
      float *dev_E;
      hipEvent_t start, stop;
      hipEventCreate(&start);
      hipEventCreate(&stop);
      
      cudaCheckReturn(hipMalloc((void **) &dev_A, (1024) * (1024) * sizeof(float)));
      cudaCheckReturn(hipMalloc((void **) &dev_B, (1024) * (1024) * sizeof(float)));
      cudaCheckReturn(hipMalloc((void **) &dev_C, (1024) * (1024) * sizeof(float)));
      cudaCheckReturn(hipMalloc((void **) &dev_D, (1024) * (1024) * sizeof(float)));
      cudaCheckReturn(hipMalloc((void **) &dev_E, (1024) * (1024) * sizeof(float)));
      
      cudaCheckReturn(hipMemcpy(dev_A, A, (1024) * (1024) * sizeof(float), hipMemcpyHostToDevice));
      cudaCheckReturn(hipMemcpy(dev_B, B, (1024) * (1024) * sizeof(float), hipMemcpyHostToDevice));
      cudaCheckReturn(hipMemcpy(dev_C, C, (1024) * (1024) * sizeof(float), hipMemcpyHostToDevice));
      cudaCheckReturn(hipMemcpy(dev_D, D, (1024) * (1024) * sizeof(float), hipMemcpyHostToDevice));
      cudaCheckReturn(hipMemcpy(dev_E, E, (1024) * (1024) * sizeof(float), hipMemcpyHostToDevice));
      {
        dim3 k0_dimBlock(16, 32);
        dim3 k0_dimGrid(32, 32);
        hipEventRecord(start,0);
        kernel0 <<<k0_dimGrid, k0_dimBlock>>> (dev_A, dev_B, dev_C);
        cudaCheckKernel();
      }
      
      {
        dim3 k1_dimBlock(16, 32);
        dim3 k1_dimGrid(32, 32);
        kernel1 <<<k1_dimGrid, k1_dimBlock>>> (dev_C, dev_D, dev_E);
        hipEventRecord(stop,0);
        hipEventSynchronize(stop);
        cudaCheckKernel();
        hipEventElapsedTime(&milliseconds, start, stop);
      }
      
      cudaCheckReturn(hipMemcpy(C, dev_C, (1024) * (1024) * sizeof(float), hipMemcpyDeviceToHost));
      cudaCheckReturn(hipMemcpy(D, dev_D, (1024) * (1024) * sizeof(float), hipMemcpyDeviceToHost));
      cudaCheckReturn(hipFree(dev_A));
      cudaCheckReturn(hipFree(dev_B));
      cudaCheckReturn(hipFree(dev_C));
      cudaCheckReturn(hipFree(dev_D));
      cudaCheckReturn(hipFree(dev_E));
    }

    
    clock_gettime(CLOCK_MONOTONIC, &finish);
    elapsed = (finish.tv_sec - start.tv_sec);
    elapsed += (finish.tv_nsec - start.tv_nsec) / 1000000000.0;
    printf("%.10f\n", elapsed);
    printf("%.10f\n", milliseconds/1000);

    return 0;
}
