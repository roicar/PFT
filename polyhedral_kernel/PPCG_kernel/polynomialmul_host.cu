#include <assert.h>
#include <stdio.h>
#include "polynomialmul_kernel.hu"
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <sys/time.h>
#include <omp.h>
#include <hip/hip_runtime.h>
#include <iostream>

#define Smax 1
#define Smin 0
#define alpha 0.6


void Init_Matrix_rand_PSxM(float matrix[1024][1024], int PS1, int PS2) {
    for (int i = 0; i < PS1; i++) {
        for (int j = 0; j < PS2; j++) {
            matrix[i][j] = (float)rand() / 32767 * (Smax - Smin) + Smin;
        }
    }
}
void Init_array_rand_PSxM(float matrix[1024], int PS1) {
    for (int i = 0; i < PS1; i++) {
        matrix[i] = (float)rand() / 32767 * (Smax - Smin) + Smin;
    }
}

void Init_Matrix_zero_PSxM(float matrix[1024][1024], int PS1, int PS2) {
    for (int i = 0; i < PS1; i++) {
        for (int j = 0; j < PS2; j++) {
            matrix[i][j] = 0;
        }
    }
}

void Init_array_zero_PSxM(float matrix[1024], int PS1) {
    for (int i = 0; i < PS1; i++) {
        matrix[i] = 0;
    }
}


int main(){
    float a[1024],b[1024],result[2049];
    double Total_time = 0.0;
    srand((unsigned)time(NULL));
    
    Init_array_rand_PSxM(a,1024);
    Init_array_rand_PSxM(b,1024);
    Init_array_zero_PSxM(result,2049);

    struct timespec start, finish;
    double elapsed;
    float milliseconds = 0;
    clock_gettime(CLOCK_MONOTONIC, &start);


    {
#define cudaCheckReturn(ret) \
  do { \
    hipError_t cudaCheckReturn_e = (ret); \
    if (cudaCheckReturn_e != hipSuccess) { \
      fprintf(stderr, "CUDA error: %s\n", hipGetErrorString(cudaCheckReturn_e)); \
      fflush(stderr); \
    } \
    assert(cudaCheckReturn_e == hipSuccess); \
  } while(0)
#define cudaCheckKernel() \
  do { \
    cudaCheckReturn(hipGetLastError()); \
  } while(0)

      float *dev_a;
      float *dev_b;
      float *dev_result;
      hipEvent_t start, stop;
      hipEventCreate(&start);
      hipEventCreate(&stop);
      
      cudaCheckReturn(hipMalloc((void **) &dev_a, (1024) * sizeof(float)));
      cudaCheckReturn(hipMalloc((void **) &dev_b, (1024) * sizeof(float)));
      cudaCheckReturn(hipMalloc((void **) &dev_result, (2049) * sizeof(float)));
      
      cudaCheckReturn(hipMemcpy(dev_a, a, (1024) * sizeof(float), hipMemcpyHostToDevice));
      cudaCheckReturn(hipMemcpy(dev_b, b, (1024) * sizeof(float), hipMemcpyHostToDevice));
      {
        dim3 k0_dimBlock(32);
        dim3 k0_dimGrid(65);
        hipEventRecord(start,0);
        kernel0 <<<k0_dimGrid, k0_dimBlock>>> (dev_a, dev_b, dev_result);
        hipEventRecord(stop,0);
        hipEventSynchronize(stop);
        cudaCheckKernel();
        hipEventElapsedTime(&milliseconds, start, stop);
      }
      
      cudaCheckReturn(hipMemcpy(result, dev_result, (2049) * sizeof(float), hipMemcpyDeviceToHost));
      cudaCheckReturn(hipFree(dev_a));
      cudaCheckReturn(hipFree(dev_b));
      cudaCheckReturn(hipFree(dev_result));
    }

    clock_gettime(CLOCK_MONOTONIC, &finish);
    elapsed = (finish.tv_sec - start.tv_sec);
    elapsed += (finish.tv_nsec - start.tv_nsec) / 1000000000.0;
    printf("%.10f\n",elapsed);
    printf("%.10f\n", milliseconds/1000);
    return 0;
}


